#include "hip/hip_runtime.h"
#include "convert_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE	32

namespace internal{

__device__ float clip(float v)
{
	if(v > 255)
		return 255;
	if(v < 0)
		return 0;
	return v;
}

__device__ float3 getRgb(float3 yuv)
{
	float3 vec;

	vec.x = yuv.x + 1.402 * (yuv.z - 128);
	vec.y = yuv.x - 0.344 * (yuv.y - 128) - 0.714 * (yuv.z - 128);
	vec.z = yuv.x + 1.772 * (yuv.y - 128);

	vec.x = clip(vec.x);
	vec.y = clip(vec.y);
	vec.z = clip(vec.z);
	return vec;
}


__global__ void convert(const uint8_t* Y, int lsY, const uint8_t* U, int lsU, const uint8_t* V, int lsV,
							uint8_t *Rgb, int lsRgb, int width, int height)
{
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	if(row >= height || col >= width)
		return;


	int y2 = (int)(row/2.);
	int x2 = (int)(col/2.);

	float3 _rgb, yuv;

	yuv.x = Y[lsY * row + col];
	yuv.y = U[lsU * y2 + x2];
	yuv.z = V[lsV * y2 + x2];

	_rgb = getRgb(yuv);

	Rgb[lsRgb * row + col * 3 + 2] = _rgb.x;
	Rgb[lsRgb * row + col * 3 + 1] = _rgb.y;
	Rgb[lsRgb * row + col * 3 + 0] = _rgb.z;
}

}

extern "C"
void convert_yuv(const uint8_t* Y, int lsY, const uint8_t* U, int lsU, const uint8_t* V, int lsV,
				 uint8_t *Rgb, int lsRgb, int width, int height)
{
	int x1 = width / BLOCKSIZE + 1;
	int x2 = height / BLOCKSIZE + 1;

	dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE, BLOCKSIZE);

	internal::convert<<<dimGrid, dimBlock>>>(Y, lsY, U, lsU, V, lsV, Rgb, lsRgb, width, height);
}

